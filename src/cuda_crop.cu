#include "hip/hip_runtime.h"
//
// (c) Bit Parallel Ltd, January 2023
//

#include "cuda_utils.hpp"
#include "cuda_crop.hpp"

template<typename T>
__global__ void gpuCrop(T* input, T* output, const int32_t offsetX, const int32_t offsetY, const int32_t inputWidth, const int32_t outputWidth, const int32_t outputHeight)
{
    const int32_t outputX = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t outputY = blockIdx.y * blockDim.y + threadIdx.y;
    if (outputX >= outputWidth || outputY >= outputHeight) return;

    const int32_t inputX = outputX + offsetX;
    const int32_t inputY = outputY + offsetY;
    output[(outputY * outputWidth) + outputX] = input[(inputY * inputWidth) + inputX];
}

template <class PixelType>
CudaCrop<PixelType>::CudaCrop(const cv::Rect inputRect, const cv::Rect outputRect):
    inputRect(inputRect), outputRect(outputRect),
    blockDim(dim3(8, 8)), gridDim(dim3(iDivRoundUp(outputRect.width, blockDim.x), iDivRoundUp(outputRect.height, blockDim.y))) {
        CUDA_CHECK(hipStreamCreate(&stream));
        init();
}

template <class PixelType>
CudaCrop<PixelType>::CudaCrop(const cv::Rect inputRect, const cv::Rect outputRect, const hipStream_t stream):
    inputRect(inputRect), outputRect(outputRect),
    blockDim(dim3(8, 8)), gridDim(dim3(iDivRoundUp(outputRect.width, blockDim.x), iDivRoundUp(outputRect.height, blockDim.y))),
    stream(stream) {
        init();
}

template <class PixelType>
void CudaCrop<PixelType>::init()
{
    CUDA_CHECK(hipMallocManaged((void**)&input, inputRect.width * inputRect.height * sizeof(PixelType)));
    CUDA_CHECK(hipMallocManaged((void**)&output, outputRect.width * outputRect.height * sizeof(PixelType)));
    CUDA_CHECK(hipEventCreate(&timerStart));
    CUDA_CHECK(hipEventCreate(&timerStop));
}

template <class PixelType>
PixelType* CudaCrop<PixelType>::getInputBuffer() const
{
    return input;
}

template <class PixelType>
PixelType* CudaCrop<PixelType>::getOutputBuffer() const
{
    return output;
}

template <class PixelType>
float CudaCrop<PixelType>::execute() const
{
    CUDA_CHECK(hipEventRecord(timerStart, stream));
    gpuCrop<PixelType><<<gridDim, blockDim, 0, stream>>>(input, output, outputRect.x, outputRect.y, inputRect.width, outputRect.width, outputRect.height);
    CUDA_CHECK(hipEventRecord(timerStop, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    float elapsedTime;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime, timerStart, timerStop));
    CUDA_CHECK(hipGetLastError());

    return elapsedTime;
}

template <class PixelType>
hipStream_t CudaCrop<PixelType>::getStream() const
{
    return stream;
}

template <class PixelType>
CudaCrop<PixelType>::~CudaCrop()
{
    // FIXME! not checking the return values
    //        problems should be reported, but there's not much that can be done
    //
    hipStreamDestroy(stream);
    hipFree(input);
    hipFree(output);

    hipEventDestroy(timerStart);
    hipEventDestroy(timerStop);
}

template <class PixelType>
int32_t CudaCrop<PixelType>::iDivRoundUp(const int32_t x, const int32_t y) const
{
    return ((x % y) != 0) ? ((x / y) + 1) : (x / y);
}

// forward references as required by non-header template classes
// add others as required
//
template class CudaCrop<uchar3>;
